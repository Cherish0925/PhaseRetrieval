#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 9 May 2020
 *      Author: Cristian Bourceanu
 */
#include "kernels.h"


__global__
void Decomp_kernel(hipfftComplex *d_signal,float *d_amp,float *d_phase,unsigned int dim){
	unsigned int index=threadIdx.x+blockIdx.x*blockDim.x;
	if(index<dim){
		d_amp[index]=sqrt(d_signal[index].x*d_signal[index].x+d_signal[index].y*d_signal[index].y);
		d_phase[index]=atan2f(d_signal[index].y,d_signal[index].x);
	}
}

__global__
void Comp_kernel(hipfftComplex *d_signal,float *d_amp,float *d_phase,unsigned int dim){
	unsigned int index=threadIdx.x+blockIdx.x*blockDim.x;
	if(index<dim){
		d_signal[index].x=d_amp[index]*cos(d_phase[index]);
		d_signal[index].y=d_amp[index]*sin(d_phase[index]);
	}
}

__global__
void amplitudeToIntensity_kernel(float *d_amp, float *d_int,unsigned int dim){
	unsigned int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<dim)
        d_int[index]=d_amp[index]*d_amp[index];
}

__global__
void scaleFourier_kernel(hipfftComplex *d_signal, unsigned int dim){
	unsigned int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<dim){
		d_signal[index].x=d_signal[index].x/dim;
		d_signal[index].y=d_signal[index].y/dim;
	}
}

__global__
void scaleVect_kernel(float *d_signal, unsigned int dim,float scale_factor){
	unsigned int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<dim)
		d_signal[index]*=scale_factor;
}

__global__
void addFloatArray_kernel(float *d_signal, unsigned int dim,float add_factor){
	unsigned int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<dim)
		d_signal[index]+=add_factor;
}

__global__
void weight_kernel(float *d_w, float *d_ampOut_before, float *d_inOut,float *d_din, unsigned int *d_ROI,unsigned int n_ROI){
    unsigned int index=threadIdx.x+blockIdx.x*blockDim.x;
    if(index<n_ROI){
        unsigned int index_ROI=d_ROI[index];
        d_w[index_ROI]=sqrtf(d_din[index_ROI]/d_inOut[index_ROI])*d_ampOut_before[index_ROI];
    }
    __syncthreads();
}

__global__
void minmax_kernel(float *d_signal,float *d_min, float *d_max,int *mutex, unsigned int length){
    __shared__ float mindata[1024];
    __shared__ float maxdata[1024];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x*gridDim.x;
    unsigned int offset=0;

    float mintemp = HIP_INF_F;
    float maxtemp = -HIP_INF_F;
    if(index==0){              //Set d_max to infinity only once to avoid racing condition
        *d_min=HIP_INF_F;
        *d_max=-HIP_INF_F;
    }
    while(index+offset<length){
        mintemp=fminf(mintemp,d_signal[index+offset]);
        maxtemp=fmaxf(maxtemp,d_signal[index+offset]);
        offset+=stride;
    }

    mindata[tid] = mintemp;
    maxdata[tid] = maxtemp;
    __syncthreads();
    

    for(unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tid<s){
            mindata[tid]=fminf(mindata[tid],mindata[tid+s]);
            maxdata[tid]=fmaxf(maxdata[tid],maxdata[tid+s]);
        }    
        __syncthreads();
    }
    
    if(tid == 0){
        while(atomicCAS(mutex,0,1));
        *d_min = fminf(*d_min,mindata[0]);
        *d_max = fmaxf(*d_max,maxdata[0]);
        atomicExch(mutex,0);
    }
}

__global__
void minmaxROI_kernel(float *d_signal,float *d_min, float *d_max,int *mutex,unsigned int *ROI, unsigned int nROI){
    __shared__ float mindata[1024];
    __shared__ float maxdata[1024];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x*gridDim.x;
    unsigned int offset=0;

    float mintemp = HIP_INF_F;
    float maxtemp = -HIP_INF_F;
    if(index==0){              //Set d_max to infinity only once to avoid racing condition
        *d_min=HIP_INF_F;
        *d_max=-HIP_INF_F;
    }
    while(index+offset<nROI){
        mintemp=fminf(mintemp,d_signal[ROI[index+offset]]);
        maxtemp=fmaxf(maxtemp,d_signal[ROI[index+offset]]);
        offset+=stride;
    }

    mindata[tid] = mintemp;
    maxdata[tid] = maxtemp;
    __syncthreads();
    
    
    if(index<nROI)
    for(unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tid<s){
            mindata[tid]=fminf(mindata[tid],mindata[tid+s]);
            maxdata[tid]=fmaxf(maxdata[tid],maxdata[tid+s]);
        }    
        __syncthreads();
    }
    
    if(tid == 0){
        while(atomicCAS(mutex,0,1));
        *d_min = fminf(*d_min,mindata[0]);
        *d_max = fmaxf(*d_max,maxdata[0]);
        atomicExch(mutex,0);
    }
    __syncthreads();
}

__global__
void sumROI_kernel(float *d_signal,float *d_sum,int *mutex,unsigned int *ROI, unsigned int nROI){
    __shared__ float data[1024];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x*gridDim.x;
    unsigned int offset=0;

    float temp = 0;
    if(index==0){              //Set d_max to infinity only once to avoid racing condition
        *d_sum=0;
    }
    while(index+offset<nROI){
        temp+=fabsf(d_signal[ROI[index+offset]]);
        offset+=stride;
    }

    data[tid] = temp;
    __syncthreads();
    
    
    if(index<nROI)
    for(unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tid<s)
            data[tid]=data[tid]+data[tid+s];  
        __syncthreads();
    }
    
    if(tid == 0){
        while(atomicCAS(mutex,0,1));
        *d_sum += data[0];
        atomicExch(mutex,0);
    }
}

__global__
void efficiency_kernel(float *d_signal,float *d_sumSR,float *d_sum,int *mutex,unsigned int *ROI, unsigned int nROI, unsigned int length){
    __shared__ float data[1024];
    __shared__ float dataSR[1024];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x*gridDim.x;
    unsigned int offset=0;

    float temp = 0;
    float tempSR = 0;
    if(index==0){              //Set d_max to infinity only once to avoid racing condition
        *d_sum=0;
        *d_sumSR=0;
    }
    while(index+offset<length){
        
        temp+=d_signal[index+offset];
        if(index+offset<nROI)
            tempSR+=d_signal[ROI[index+offset]];
        offset+=stride;
    }

    data[tid] = temp;
    dataSR[tid] = tempSR;
    __syncthreads();
    
    
    for(unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tid<s){
            data[tid]=data[tid]+data[tid+s];
            dataSR[tid]=dataSR[tid]+dataSR[tid+s];   
        }
        __syncthreads();
    }
    
    if(tid == 0){
        while(atomicCAS(mutex,0,1));
        *d_sum += data[0];
        *d_sumSR+=dataSR[0];
        atomicExch(mutex,0);
    }
}

__global__
void accuracy_kernel(float *d_iOut,float *d_di,float *d_sumerr2,int *mutex,unsigned int *ROI, unsigned int nROI){
    __shared__ float dataerr[256];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x*gridDim.x;
    unsigned int indexROI;
    unsigned int offset=0;

    float temperr = 0;
    if(index==0){            //Set d_max only once to avoid racing condition
        *d_sumerr2=0;
    }
    
    while(index+offset<nROI){
        indexROI=ROI[index+offset];
        temperr+=fdividef(powf(d_iOut[indexROI]-d_di[indexROI],2),powf(d_di[indexROI],2));
        offset+=stride;
    }
    dataerr[tid] = temperr;
    __syncthreads();
    
    
    for(unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tid<s){
            dataerr[tid]+=dataerr[tid+s]; 
        }
        __syncthreads();
    }
    
    if(tid == 0){
        while(atomicCAS(mutex,0,1));
        __threadfence();
        *d_sumerr2 += dataerr[0];
        atomicExch(mutex,0);
    }
}

__global__
void addROI_kernel(float *d_in,float scale_in,float *d_out,float scale_out,unsigned int *ROI, unsigned int nROI){
    unsigned int index=threadIdx.x+blockIdx.x*blockDim.x;
    if(index<nROI){
        unsigned int index_ROI=ROI[index];
        d_out[index_ROI]=scale_in*d_in[index_ROI]+scale_out*d_out[index_ROI];
    }
    __syncthreads();
}

__global__
void sqrt_kernel(float *d_signal_in,float *d_signal_out,unsigned int length){
    unsigned int index=threadIdx.x+blockIdx.x*blockDim.x;

    if(index<length){
        d_signal_out[index]=sqrtf(d_signal_in[index]);
    }
    __syncthreads();
}
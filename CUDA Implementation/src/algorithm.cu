#include "hip/hip_runtime.h"
/*
 * algorithm.cu
 *
 *  Created on: 11 May 2020
 *      Author: cristi
 */
#include "algorithm.h"


//*********** OppBlocks Definitions ***********//

OppBlocks::OppBlocks(int nx,int ny):nx(nx),ny(ny){
	if((error = hipfftPlan2d(&planFFT,nx,ny, HIPFFT_C2C))!=HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
	}
	if((stat_cublas = hipblasCreate(&handle_cublas))!=HIPBLAS_STATUS_SUCCESS){
		printf("cuBLAS error: Handle creation failed");
	}
	if((stat_curand =hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS){
		printf("cuRAND error: Generator creation failed");
	}
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL));
	CUDA_CALL(hipMalloc((void**)&d_min,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_max,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_mutex,sizeof(int)));
}
OppBlocks::~OppBlocks(){
	hipFree(d_min);
	hipFree(d_max);
	hipFree(d_mutex);
	hipfftDestroy(planFFT);
	hipfftDestroy(planFFT);
	CUBLAS_CALL(hipblasDestroy(handle_cublas)); 
	printf("OppBlocks destructed successfully!\n");
}
void OppBlocks::SLM_To_Obj(hipComplex *d_SLM,hipComplex *d_Obj){
	hipfftExecC2C(planFFT,d_SLM,d_Obj,HIPFFT_BACKWARD);
	//scaleFourier_kernel<<<(nx*ny+1023)/1024,1024>>>(d_Obj,nx*ny);
	float scale = 1.0/(nx*ny);
	hipDeviceSynchronize();
	hipblasCsscal(handle_cublas,nx*ny,&scale,d_Obj,1);

}
void OppBlocks::Obj_to_SLM(hipComplex *d_Obj,hipComplex *d_SLM){
	hipfftExecC2C(planFFT,d_Obj,d_SLM,HIPFFT_FORWARD);
}

void OppBlocks::Compose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Comp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OppBlocks::Decompose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Decomp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OppBlocks::RandomArray(float* d_array,float min, float max){
	hiprandGenerateNormal(curand_gen,d_array,nx*ny,min,max);
}
void OppBlocks::Normalize(float *d_quantity){
	float h_min,h_max;

	max_kernel<<<32,1024>>>(d_quantity,d_max,d_mutex,nx*ny);
	hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost);
	min_kernel<<<32,1024>>>(d_quantity,d_min,d_mutex,nx*ny);
	hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost);
	float scale=1/(h_max-h_min);
	hipblasSscal(handle_cublas,nx*ny,&scale,d_quantity,1);
	hipDeviceSynchronize();
	//scaleAmp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,nx*ny,h_max-h_min);
	addFloatArray_kernel<<<(nx*ny+1023)/1024,1024>>>(d_quantity,nx*ny,-h_min/(h_max-h_min));		//Couldn't find cublas to add scalar to an array
	printf("(min,max)=(%f,%f)\n",h_min,h_max);
}
void OppBlocks::NormalizedIntensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
	Normalize(d_intensity);
}




//*********** PhaseRetrieve ***********//

PhaseRetrieve::PhaseRetrieve(float *gray_img,int nx, int ny, PR_Type type):OppBlocks(nx,ny){
	InitGPU(0);

	//Host memory allocation
	host->complex=(hipComplex*)malloc(nx*ny*sizeof(hipComplex));
	host->illum=(float*)malloc(nx*ny*sizeof(float));
	host->damp=(float*)malloc(nx*ny*sizeof(float));
	host->amp=(float*)malloc(nx*ny*sizeof(float));
	host->phase=(float*)malloc(nx*ny*sizeof(float));
	host->intensity=(float*)malloc(nx*ny*sizeof(float));
	h_out_img=(float*)malloc(nx*ny*sizeof(float));
	h_out_phase=(float*)malloc(nx*ny*sizeof(float));

	//Device memory allocation
	CUDA_CALL(hipMalloc((void**)&device->complex,nx*ny*sizeof(hipComplex)));
	CUDA_CALL(hipMalloc((void**)&device->illum,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->damp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->amp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->phase,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->intensity,nx*ny*sizeof(float)));

	SetImage(gray_img);
	SetIllumination();
}

PhaseRetrieve::~PhaseRetrieve(){
	free(host->complex);	free(host->damp);		free(host->amp);		free(host->phase);
	free(host->intensity);		free(h_out_img);	free(h_out_phase);	free(host->illum);
	hipFree(device->complex);	hipFree(device->damp);
	hipFree(device->amp);		hipFree(device->phase);	
	hipFree(device->intensity);		hipFree(device->illum);
	printf("PhaseRetrieve destructed successfully!\n");
}
void PhaseRetrieve::InitGPU(int device_id){
	int devCount;
    hipGetDeviceCount(&devCount);	//number of GPUs available
	if(device_id<devCount)		//check if there are enogh GPUs
        hipSetDevice(device_id);
    else exit(1);
}
void PhaseRetrieve::SetImage(float *gray_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->damp[index(i,j)]=sqrt(gray_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device->damp,host->damp,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(float *illum_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->illum[index(i,j)]=sqrt(illum_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device->illum,host->illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->illum[index(i,j)]=sqrt(255);
	CUDA_CALL(hipMemcpy(device->illum,host->illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
unsigned int PhaseRetrieve::index(unsigned int i, unsigned int j){
	return nx*i+j;
}
void PhaseRetrieve::Test(){

	RandomArray(device->phase,-M_PI,M_PI);

	for(int i=0;i<50;i++){
		Compose(device->complex,device->damp,device->phase);
		Obj_to_SLM(device->complex,device->complex);
		Decompose(device->complex,device->amp,device->phase);
		Compose(device->complex,device->illum,device->phase);
		SLM_To_Obj(device->complex,device->complex);
		Decompose(device->complex,device->amp,device->phase);
	}

	NormalizedIntensity(device->amp,device->intensity);

	CUDA_CALL(hipMemcpy(host->amp,device->amp,nx*ny*sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(host->intensity,device->intensity,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	Obj_to_SLM(device->complex,device->complex);
	Decompose(device->complex,device->amp,device->phase);

	Normalize(device->phase);
	CUDA_CALL(hipMemcpy(h_out_phase,device->phase,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	float err=0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			err+=pow((host->damp[index(i,j)]-host->amp[index(i,j)]),2);
			h_out_img[index(i,j)]=255*host->intensity[index(i,j)];
			h_out_phase[index(i,j)]=255*h_out_phase[index(i,j)];
		}

	printf("Error squared: %f\n",err);
}

float* PhaseRetrieve::GetImage(){
	return h_out_img;
}

float* PhaseRetrieve::GetPhaseMask(){
	return h_out_phase;
}
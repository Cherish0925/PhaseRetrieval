#include "hip/hip_runtime.h"
/*
 * algorithm.cu
 *
 *  Created on: 6 May 2020
 *      Author: Cristian Bourceanu
 */
#include "algorithm.h"


//*********** OpBlocks Definitions ***********//

OpBlocks::OpBlocks(int nx,int ny):nx(nx),ny(ny){
	if((error = hipfftPlan2d(&planFFT,nx,ny, HIPFFT_C2C))!=HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
	}
	if((stat_cublas = hipblasCreate(&handle_cublas))!=HIPBLAS_STATUS_SUCCESS){
		printf("cuBLAS error: Handle creation failed");
	}
	if((stat_curand =hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS){
		printf("cuRAND error: Generator creation failed");
	}
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL));
	CUDA_CALL(hipMalloc((void**)&d_min,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_max,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_mutex,sizeof(int)));
}
OpBlocks::~OpBlocks(){
	CUDA_CALL(hipFree(d_min));
	CUDA_CALL(hipFree(d_max));
	CUDA_CALL(hipFree(d_mutex));
	CUFFT_CALL(hipfftDestroy(planFFT));
	CUBLAS_CALL(hipblasDestroy(handle_cublas)); 
	printf("OpBlocks destructed successfully!\n");
}
void OpBlocks::SLM_To_Obj(hipComplex *d_SLM,hipComplex *d_Obj){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_SLM,d_Obj,HIPFFT_BACKWARD));
	//scaleFourier_kernel<<<(nx*ny+1023)/1024,1024>>>(d_Obj,nx*ny);
	float scale = 1.0/(nx*ny);
	hipDeviceSynchronize();
	CUBLAS_CALL(hipblasCsscal(handle_cublas,nx*ny,&scale,d_Obj,1));

}
void OpBlocks::Obj_to_SLM(hipComplex *d_Obj,hipComplex *d_SLM){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_Obj,d_SLM,HIPFFT_FORWARD));
}

void OpBlocks::Compose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Comp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::Decompose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Decomp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::RandomArray(float* d_array,float min, float max){
	hiprandGenerateNormal(curand_gen,d_array,nx*ny,min,max);
}
void OpBlocks::Normalize(float *d_quantity){
	float h_min,h_max;

	max_kernel<<<32,1024>>>(d_quantity,d_max,d_mutex,nx*ny);
	hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost);
	min_kernel<<<32,1024>>>(d_quantity,d_min,d_mutex,nx*ny);
	hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost);
	float scale=1/(h_max-h_min);
	CUBLAS_CALL(hipblasSscal(handle_cublas,nx*ny,&scale,d_quantity,1));
	hipDeviceSynchronize();
	//scaleAmp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,nx*ny,h_max-h_min);
	addFloatArray_kernel<<<(nx*ny+1023)/1024,1024>>>(d_quantity,nx*ny,-h_min/(h_max-h_min));		//Couldn't find cublas to add scalar to an array
	printf("(min,max)=(%f,%f)\n",h_min,h_max);
}
void OpBlocks::NormalizedIntensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
	Normalize(d_intensity);
}




//*********** PhaseRetrieve ***********//

PhaseRetrieve::PhaseRetrieve(float *gray_img,unsigned int nx,unsigned int ny, PR_Type type):
nx(nx),ny(ny){
	InitGPU(0);

	//Host memory allocation
	host->complex=(hipComplex*)malloc(nx*ny*sizeof(hipComplex));
	host->illum=(float*)malloc(nx*ny*sizeof(float));
	host->damp=(float*)malloc(nx*ny*sizeof(float));
	host->amp=(float*)malloc(nx*ny*sizeof(float));
	host->phase=(float*)malloc(nx*ny*sizeof(float));
	host->intensity=(float*)malloc(nx*ny*sizeof(float));
	h_out_img=(float*)malloc(nx*ny*sizeof(float));
	h_out_phase=(float*)malloc(nx*ny*sizeof(float));

	//Device memory allocation
	CUDA_CALL(hipMalloc((void**)&device->complex,nx*ny*sizeof(hipComplex)));
	CUDA_CALL(hipMalloc((void**)&device->illum,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->damp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->amp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->phase,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device->intensity,nx*ny*sizeof(float)));

	SetImage(gray_img);
	SetIllumination();
	operation=new OpBlocks(nx,ny);
	SetAlgorithm(type);
}

PhaseRetrieve::~PhaseRetrieve(){
	free(host->complex);	free(host->damp);		free(host->amp);		free(host->phase);
	free(host->intensity);		free(h_out_img);	free(h_out_phase);	free(host->illum);
	hipFree(device->complex);	hipFree(device->damp);
	hipFree(device->amp);		hipFree(device->phase);	
	hipFree(device->intensity);		hipFree(device->illum);
	delete[] device;
	delete[] host;
	delete operation;
	delete algorithm;
	printf("PhaseRetrieve destructed successfully!\n");
}
void PhaseRetrieve::InitGPU(int device_id){
	int devCount;
    hipGetDeviceCount(&devCount);	//number of GPUs available
	if(device_id<devCount)		//check if there are enogh GPUs
        hipSetDevice(device_id);
    else exit(1);
}
void PhaseRetrieve::SetAlgorithm(PR_Type type){
	if(algorithm)
		delete algorithm;
	algorithm=AlgorithmCreator().FactoryMethod(operation,device,host,type);
}
void PhaseRetrieve::SetImage(float *gray_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->damp[index(i,j)]=sqrt(gray_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device->damp,host->damp,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(float *illum_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->illum[index(i,j)]=sqrt(illum_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device->illum,host->illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host->illum[index(i,j)]=sqrt(255);
	CUDA_CALL(hipMemcpy(device->illum,host->illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
unsigned int PhaseRetrieve::index(unsigned int i, unsigned int j){
	return nx*i+j;
}
void PhaseRetrieve::Test(){

	operation->RandomArray(device->phase,-M_PI,M_PI);

	for(int i=0;i<1000;i++){
		algorithm->OneIteration();
		operation->Decompose(device->complex,device->amp,device->phase);
		operation->Compose(device->complex,device->illum,device->phase);
		operation->SLM_To_Obj(device->complex,device->complex);
		operation->Decompose(device->complex,device->amp,device->phase);
	}

	operation->NormalizedIntensity(device->amp,device->intensity);

	CUDA_CALL(hipMemcpy(host->amp,device->amp,nx*ny*sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(host->intensity,device->intensity,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Obj_to_SLM(device->complex,device->complex);
	operation->Decompose(device->complex,device->amp,device->phase);

	operation->Normalize(device->phase);
	CUDA_CALL(hipMemcpy(h_out_phase,device->phase,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	float err=0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			err+=pow((host->damp[index(i,j)]-host->amp[index(i,j)]),2);
			h_out_img[index(i,j)]=255*host->intensity[index(i,j)];
			h_out_phase[index(i,j)]=255*h_out_phase[index(i,j)];
		}

	printf("Error squared: %f\n",err);
}

float* PhaseRetrieve::GetImage(){
	return h_out_img;
}

float* PhaseRetrieve::GetPhaseMask(){
	return h_out_phase;
}



/********** Algorithms Implementation ***************/

void GS_ALG::OneIteration(){
	operation->Compose(device->complex,device->damp,device->phase);
	operation->Obj_to_SLM(device->complex,device->complex);
}
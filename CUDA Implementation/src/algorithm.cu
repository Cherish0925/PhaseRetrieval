#include "hip/hip_runtime.h"
/*
 * algorithm.cu
 *
 *  Created on: 11 May 2020
 *      Author: cristi
 */
#include "algorithm.h"


//*********** OppBlocks Definitions ***********//

OppBlocks::OppBlocks(int nx,int ny):nx(nx),ny(ny){
	if((error = hipfftPlan2d(&planFFT,nx,ny, HIPFFT_C2C))!=HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
	}
	if((stat_cublas = hipblasCreate(&handle_cublas))!=HIPBLAS_STATUS_SUCCESS){
		printf("cuBLAS error: Handle creation failed");
	}
	if((stat_curand =hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS){
		printf("cuRAND error: Generator creation failed");
	}
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL));
	CUDA_CALL(hipMalloc((void**)&d_min,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_max,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_mutex,sizeof(int)));
}
OppBlocks::~OppBlocks(){
	hipFree(d_min);
	hipFree(d_max);
	hipFree(d_mutex);
	hipfftDestroy(planFFT);
	hipfftDestroy(planFFT);
	CUBLAS_CALL(hipblasDestroy(handle_cublas)); 
	printf("OppBlocks destructed successfully!\n");
}
void OppBlocks::SLM_To_Obj(hipComplex *d_SLM,hipComplex *d_Obj){
	hipfftExecC2C(planFFT,d_SLM,d_Obj,HIPFFT_BACKWARD);
	//scaleFourier_kernel<<<(nx*ny+1023)/1024,1024>>>(d_Obj,nx*ny);
	float scale = 1.0/(nx*ny);
	hipDeviceSynchronize();
	hipblasCsscal(handle_cublas,nx*ny,&scale,d_Obj,1);

}
void OppBlocks::Obj_to_SLM(hipComplex *d_Obj,hipComplex *d_SLM){
	hipfftExecC2C(planFFT,d_Obj,d_SLM,HIPFFT_FORWARD);
}

void OppBlocks::Compose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Comp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OppBlocks::Decompose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Decomp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OppBlocks::RandomArray(float* d_array,float min, float max){
	hiprandGenerateNormal(curand_gen,d_array,nx*ny,min,max);
}
void OppBlocks::Normalize(float *d_quantity){
	float h_min,h_max;

	max_kernel<<<32,1024>>>(d_quantity,d_max,d_mutex,nx*ny);
	hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost);
	min_kernel<<<32,1024>>>(d_quantity,d_min,d_mutex,nx*ny);
	hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost);
	float scale=1/(h_max-h_min);
	hipblasSscal(handle_cublas,nx*ny,&scale,d_quantity,1);
	hipDeviceSynchronize();
	//scaleAmp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,nx*ny,h_max-h_min);
	addFloatArray_kernel<<<(nx*ny+1023)/1024,1024>>>(d_quantity,nx*ny,-h_min/(h_max-h_min));		//Couldn't find cublas to add scalar to an array
	printf("(min,max)=(%f,%f)\n",h_min,h_max);
}
void OppBlocks::NormalizedIntensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
	Normalize(d_intensity);
}




//*********** PhaseRetrieve ***********//

PhaseRetrieve::PhaseRetrieve(float *gray_img,int nx, int ny, PR_Type type):OppBlocks(nx,ny){
	InitGPU(0);

	//Host memory allocation
	h_complex=(hipComplex*)malloc(nx*ny*sizeof(hipComplex));
	h_illum=(float*)malloc(nx*ny*sizeof(float));
	h_damp=(float*)malloc(nx*ny*sizeof(float));
	h_amp=(float*)malloc(nx*ny*sizeof(float));
	h_phase=(float*)malloc(nx*ny*sizeof(float));
	h_int=(float*)malloc(nx*ny*sizeof(float));
	h_out_img=(float*)malloc(nx*ny*sizeof(float));
	h_out_phase=(float*)malloc(nx*ny*sizeof(float));

	//Device memory allocation
	CUDA_CALL(hipMalloc((void**)&d_complex,nx*ny*sizeof(hipComplex)));
	CUDA_CALL(hipMalloc((void**)&d_illum,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_damp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_amp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_phase,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_int,nx*ny*sizeof(float)));

	SetImage(gray_img);
}

PhaseRetrieve::~PhaseRetrieve(){
	free(h_complex);	free(h_damp);		free(h_amp);		free(h_phase);
	free(h_int);		free(h_out_img);	free(h_out_phase);	free(h_illum);
	hipFree(d_complex);	hipFree(d_damp);
	hipFree(d_amp);		hipFree(d_phase);	
	hipFree(d_int);		hipFree(d_illum);
	printf("PhaseRetrieve destructed successfully!\n");
}
void PhaseRetrieve::InitGPU(int device_id){
	int devCount;
    hipGetDeviceCount(&devCount);	//number of GPUs available
	if(device_id<devCount)		//check if there are enogh GPUs
        hipSetDevice(device_id);
    else exit(1);
}
void PhaseRetrieve::SetImage(float *gray_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			h_damp[index(i,j)]=sqrt(gray_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(d_damp,h_damp,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			h_illum[index(i,j)]=sqrt(255);
	CUDA_CALL(hipMemcpy(d_illum,h_illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
unsigned int PhaseRetrieve::index(unsigned int i, unsigned int j){
	return nx*i+j;
}
void PhaseRetrieve::Test(){
	SetIllumination();

	RandomArray(d_phase,-M_PI,M_PI);

	for(int i=0;i<1000;i++){
		Compose(d_complex,d_damp,d_phase);
		Obj_to_SLM(d_complex,d_complex);
		Decompose(d_complex,d_amp,d_phase);
		Compose(d_complex,d_illum,d_phase);
		SLM_To_Obj(d_complex,d_complex);
		Decompose(d_complex,d_amp,d_phase);
	}

	NormalizedIntensity(d_amp,d_int);

	CUDA_CALL(hipMemcpy(h_amp,d_amp,nx*ny*sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(h_int,d_int,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	Normalize(d_phase);
	CUDA_CALL(hipMemcpy(h_out_phase,d_phase,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	float err=0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			err+=pow((h_damp[index(i,j)]-h_amp[index(i,j)]),2);
			h_out_img[index(i,j)]=255*h_int[index(i,j)];
			h_out_phase[index(i,j)]=255*h_out_phase[index(i,j)];
		}

	printf("Error squared: %f\n",err);
}

float* PhaseRetrieve::GetImage(){
	return h_out_img;
}

float* PhaseRetrieve::GetPhaseMask(){
	return h_out_phase;
}
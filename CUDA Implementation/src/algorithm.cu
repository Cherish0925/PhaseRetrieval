#include "hip/hip_runtime.h"
/*
 * algorithm.cu
 *
 *  Created on: 9 May 2020
 *      Author: Cristian Bourceanu
 */
#include "algorithm.h"


//*********** OpBlocks Definitions ***********//

OpBlocks::OpBlocks(int nx,int ny):nx(nx),ny(ny){
	if((error = hipfftPlan2d(&planFFT,nx,ny, HIPFFT_C2C))!=HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
	}
	if((stat_cublas = hipblasCreate(&handle_cublas))!=HIPBLAS_STATUS_SUCCESS){
		printf("cuBLAS error: Handle creation failed");
	}
	if((stat_curand =hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS){
		printf("cuRAND error: Generator creation failed");
	}
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL));
	CUDA_CALL(hipMalloc((void**)&d_min,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_max,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_mutex,sizeof(int)));
	CUDA_CALL(hipMemset(d_mutex,0,sizeof(int)));
}
OpBlocks::~OpBlocks(){
	CUDA_CALL(hipFree(d_min));
	CUDA_CALL(hipFree(d_max));
	CUDA_CALL(hipFree(d_mutex));
	CUFFT_CALL(hipfftDestroy(planFFT));
	CUBLAS_CALL(hipblasDestroy(handle_cublas)); 
	printf("OpBlocks destructed successfully!\n");
}
hipblasHandle_t& OpBlocks::GetCUBLAS(){
	return handle_cublas;
}
void OpBlocks::SLM_To_Obj(hipComplex *d_SLM,hipComplex *d_Obj){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_SLM,d_Obj,HIPFFT_BACKWARD));
	//scaleFourier_kernel<<<(nx*ny+1023)/1024,1024>>>(d_Obj,nx*ny);
	float scale = 1.0/(nx*ny);
	hipDeviceSynchronize();
	CUBLAS_CALL(hipblasCsscal(handle_cublas,nx*ny,&scale,d_Obj,1));

}
void OpBlocks::Obj_to_SLM(hipComplex *d_Obj,hipComplex *d_SLM){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_Obj,d_SLM,HIPFFT_FORWARD));
}

void OpBlocks::Compose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Comp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::Decompose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Decomp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::Sum(float *d_adto,float *d_increment){
	const float one=1.0f;
	CUBLAS_CALL(hipblasSaxpy(handle_cublas,nx*ny,&one,d_increment,1,d_adto,1));
}
void OpBlocks::Scale(float *d_signal,float scaling){
	CUBLAS_CALL(hipblasSscal(handle_cublas,nx*ny,&scaling,d_signal,1));
}
void OpBlocks::RandomArray(float* d_array,float min, float max){
	hiprandGenerateNormal(curand_gen,d_array,nx*ny,min,max);
}
void OpBlocks::ZeroArray(float* d_array,size_t n_bytes){
	CUDA_CALL(hipMemset(d_array,0,n_bytes));
}
void OpBlocks::Normalize(float *d_quantity){
	float h_min,h_max;
	minmax_kernel<<<32,1024>>>(d_quantity,d_min,d_max,d_mutex,nx*ny);
	hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost);
	float scale=1/(h_max-h_min);
	CUBLAS_CALL(hipblasSscal(handle_cublas,nx*ny,&scale,d_quantity,1));
	hipDeviceSynchronize();
	addFloatArray_kernel<<<(nx*ny+1023)/1024,1024>>>(d_quantity,nx*ny,-h_min/(h_max-h_min));		//Couldn't find cublas to add scalar to an array
	//printf("(min,max)=(%f,%f)\n",h_min,h_max);
}
void OpBlocks::Intensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
}
void OpBlocks::NormalizedIntensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
	Normalize(d_intensity);
}
/**
 * @brief Uniformity within the region of interest
 * 
 * @param d_signal Signal whose host.uniformity is assesed
 * @param d_ROI 	Array of indexes of the elements in the ROI
 * @param n_ROI 	Length of d_ROI
 */
float OpBlocks::Uniformity(float *d_signal,unsigned int *d_ROI,unsigned int n_ROI){
	minmaxROI_kernel<<<32,1024>>>(d_signal,d_min,d_max,d_mutex,d_ROI,n_ROI);
	float h_min,h_max;
	CUDA_CALL(hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost));
	return 1-(h_max-h_min)/(h_max+h_min);
}



//*********** PhaseRetrieve ***********//

PhaseRetrieve::PhaseRetrieve(float *gray_img,unsigned int nx,unsigned int ny, PR_Type type):
nx(nx),ny(ny){
	InitGPU(0);

	//Host memory allocation
	host.complex=(hipComplex*)malloc(nx*ny*sizeof(hipComplex));
	host.illum=(float*)malloc(nx*ny*sizeof(float));
	host.dint=(float*)malloc(nx*ny*sizeof(float));
	host.damp=(float*)malloc(nx*ny*sizeof(float));
	host.amp=(float*)malloc(nx*ny*sizeof(float));
	host.ampOut=(float*)malloc(nx*ny*sizeof(float));
	host.phSLM=(float*)malloc(nx*ny*sizeof(float));
	host.phImg=(float*)malloc(nx*ny*sizeof(float));
	host.intensity=(float*)malloc(nx*ny*sizeof(float));
	h_out_img=(float*)malloc(nx*ny*sizeof(float));
	h_out_phase=(float*)malloc(nx*ny*sizeof(float));

	//Device memory allocation
	CUDA_CALL(hipMalloc((void**)&device.complex,nx*ny*sizeof(hipComplex)));
	CUDA_CALL(hipMalloc((void**)&device.illum,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.dint,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.damp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.amp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.ampOut,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.phSLM,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.phImg,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.intensity,nx*ny*sizeof(float)));

	host.nx=nx;	host.ny=ny;

	SetImage(gray_img);
	SetIllumination();
	operation=new OpBlocks(nx,ny);
	SetAlgorithm(type);
}

PhaseRetrieve::~PhaseRetrieve(){
	free(host.complex);	free(host.damp);		free(host.dint);		free(host.amp);		free(host.phSLM);
	free(host.intensity);		free(h_out_img);	free(h_out_phase);	free(host.illum);
	free(host.ampOut);		free(host.phImg);
	hipFree(device.complex);	hipFree(device.damp);		hipFree(device.dint);
	hipFree(device.amp);		hipFree(device.phSLM);	
	hipFree(device.intensity);		hipFree(device.illum);
	hipFree(device.ampOut);	hipFree(device.phImg);
	if(host.ROI){	free(host.ROI);	hipFree(device.ROI);}	
	
	delete algorithm;
	delete operation;
	printf("PhaseRetrieve destructed successfully!\n");
}
void PhaseRetrieve::InitGPU(int device_id){
	int devCount;
    hipGetDeviceCount(&devCount);	//number of GPUs available
	if(device_id<devCount)		//check if there are enogh GPUs
        hipSetDevice(device_id);
    else exit(1);
}
void PhaseRetrieve::SetAlgorithm(PR_Type type){
	if(algorithm)
		delete algorithm;
	algorithm=AlgorithmCreator().FactoryMethod(operation,device,host,type);
}
void PhaseRetrieve::SetImage(float *gray_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			host.damp[index(i,j)]=sqrt(gray_img[index(i,j)]);
			host.dint[index(i,j)]=gray_img[index(i,j)];
		}
	CUDA_CALL(hipMemcpy(device.damp,host.damp,nx*ny*sizeof(float),hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(device.dint,host.dint,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(float *illum_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host.illum[index(i,j)]=sqrt(illum_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device.illum,host.illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host.illum[index(i,j)]=sqrt(255);
	CUDA_CALL(hipMemcpy(device.illum,host.illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::FindROI(float threshold){
	if(host.n_ROI==0){
		host.ROI=(unsigned int*)malloc(host.nx*host.ny*sizeof(unsigned int));
		for(unsigned int i=0;i<host.nx*host.ny;i++)
			if(host.damp[i]>threshold){
				host.ROI[host.n_ROI++]=i;
			}
		CUDA_CALL(hipMalloc((void**)&device.ROI,host.n_ROI*sizeof(unsigned int)));
		CUDA_CALL(hipMemcpy(device.ROI,host.ROI,host.n_ROI*sizeof(unsigned int),hipMemcpyHostToDevice));
	}
}
unsigned int PhaseRetrieve::index(unsigned int i, unsigned int j){
	return nx*i+j;
}
void PhaseRetrieve::Test(){
	
	operation->RandomArray(device.phImg,-M_PI,M_PI);
	operation->RandomArray(device.phSLM,-M_PI,M_PI);

	FindROI(sqrt(255)/2);
	algorithm->Initialize();
	
	for(int i=0;i<50;i++){
		//if(i==4) SetAlgorithm(MRAF);
		algorithm->OneIteration();
		// if(host.n_ROI)
		// 	host.uniformity.push_back(operation->Uniformity(device.intensity,device.ROI,host.n_ROI));
	}

	operation->NormalizedIntensity(device.ampOut,device.intensity);

	CUDA_CALL(hipMemcpy(host.intensity,device.intensity,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.ampOut);
	CUDA_CALL(hipMemcpy(host.ampOut,device.ampOut,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.phSLM);
	CUDA_CALL(hipMemcpy(h_out_phase,device.phSLM,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.damp);
	CUDA_CALL(hipMemcpy(host.damp,device.damp,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	float err=0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			err+=pow((host.damp[index(i,j)]-host.ampOut[index(i,j)]),2);
			h_out_img[index(i,j)]=255*host.intensity[index(i,j)];
			h_out_phase[index(i,j)]=255*h_out_phase[index(i,j)];
		}

	printf("Error squared: %f\n",err);
}

float* PhaseRetrieve::GetImage(){
	return h_out_img;
}

float* PhaseRetrieve::GetPhaseMask(){
	return h_out_phase;
}

std::vector<float>& PhaseRetrieve::GetUniformity(){
	return host.uniformity;
}


/********** Algorithms Implementation ***************/
void GS_ALG::Initialize(){

}
void GS_ALG::OneIteration(){
	operation->Compose(device.complex,device.damp,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
	operation->Intensity(device.ampOut,device.intensity);
}

void MRAF_ALG::Initialize(){
	//operation->ZeroArray(device.ampOut,host.nx * host.ny);
}
void MRAF_ALG::OneIteration(){
	//MRAF Scaling the desired amplitude for correction
	const float lambda = 1.0f;
	operation->Scale(device.ampOut,lambda*(host.uniformity.back()-1));
	float scale_des=1;
	CUBLAS_CALL(hipblasSaxpy(operation->GetCUBLAS(),host.nx*host.ny,&scale_des,device.damp,1,device.ampOut,1));

	operation->Compose(device.complex,device.ampOut,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
}
void WGS_ALG::Initialize(){
	//float *ampOut_before,*wamp;
}
void WGS_ALG::OneIteration(){
	//operation->Compose(device.complex,wamp,device.phImg);
	//hipMemcpy(ampOut_before,device.ampOut,host.nx*host.ny*sizeof(float),hipMemcpyDeviceToDevice);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
}
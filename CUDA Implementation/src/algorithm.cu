#include "hip/hip_runtime.h"
/*
 * algorithm.cu
 *
 *  Created on: 9 May 2020
 *      Author: Cristian Bourceanu
 */
#include "algorithm.h"


//*********** OpBlocks Definitions ***********//

OpBlocks::OpBlocks(int nx,int ny):nx(nx),ny(ny){
	if((error = hipfftPlan2d(&planFFT,nx,ny, HIPFFT_C2C))!=HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed");
	}
	if((stat_cublas = hipblasCreate(&handle_cublas))!=HIPBLAS_STATUS_SUCCESS){
		printf("cuBLAS error: Handle creation failed");
	}
	if((stat_curand =hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT))!=HIPRAND_STATUS_SUCCESS){
		printf("cuRAND error: Generator creation failed");
	}
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL));
	CUDA_CALL(hipMalloc((void**)&d_min,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_max,sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_mutex,sizeof(int)));
	CUDA_CALL(hipMemset(d_mutex,0,sizeof(int)));
}
OpBlocks::~OpBlocks(){
	CUDA_CALL(hipFree(d_min));
	CUDA_CALL(hipFree(d_max));
	CUDA_CALL(hipFree(d_mutex));
	CUFFT_CALL(hipfftDestroy(planFFT));
	CUBLAS_CALL(hipblasDestroy(handle_cublas)); 
	printf("OpBlocks destructed successfully!\n");
}
hipblasHandle_t& OpBlocks::GetCUBLAS(){
	return handle_cublas;
}
void OpBlocks::SLM_To_Obj(hipComplex *d_SLM,hipComplex *d_Obj){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_SLM,d_Obj,HIPFFT_BACKWARD));
	//scaleFourier_kernel<<<(nx*ny+1023)/1024,1024>>>(d_Obj,nx*ny);
	float scale = 1.0/(nx*ny);
	hipDeviceSynchronize();
	CUBLAS_CALL(hipblasCsscal(handle_cublas,nx*ny,&scale,d_Obj,1));

}
void OpBlocks::Obj_to_SLM(hipComplex *d_Obj,hipComplex *d_SLM){
	CUFFT_CALL(hipfftExecC2C(planFFT,d_Obj,d_SLM,HIPFFT_FORWARD));
}

void OpBlocks::Compose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Comp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::Decompose(hipComplex *d_signal,float *d_amp,float *d_phase){
	Decomp_kernel<<<(nx*ny+1023)/1024,1024>>>(d_signal,d_amp,d_phase,nx*ny);
}
void OpBlocks::Sum(float *d_adto,float *d_increment){
	const float one=1.0f;
	CUBLAS_CALL(hipblasSaxpy(handle_cublas,nx*ny,&one,d_increment,1,d_adto,1));
}
void OpBlocks::Scale(float *d_signal,float scaling){
	CUBLAS_CALL(hipblasSscal(handle_cublas,nx*ny,&scaling,d_signal,1));
}
void OpBlocks::RandomArray(float* d_array,float min, float max){
	hiprandGenerateNormal(curand_gen,d_array,nx*ny,min,max);
}
void OpBlocks::ZeroArray(float* d_array,size_t n_bytes){
	CUDA_CALL(hipMemset(d_array,0,n_bytes));
}
void OpBlocks::Normalize(float *d_quantity){
	float h_min,h_max;
	minmax_kernel<<<32,1024>>>(d_quantity,d_min,d_max,d_mutex,nx*ny);
	hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost);
	float scale=1/(h_max-h_min);
	CUBLAS_CALL(hipblasSscal(handle_cublas,nx*ny,&scale,d_quantity,1));
	hipDeviceSynchronize();
	addFloatArray_kernel<<<(nx*ny+1023)/1024,1024>>>(d_quantity,nx*ny,-h_min/(h_max-h_min));		//Couldn't find cublas to add scalar to an array
	//printf("(min,max)=(%f,%f)\n",h_min,h_max);
}
void OpBlocks::Intensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
}
void OpBlocks::NormalizedIntensity(float *d_amp,float *d_intensity){
	amplitudeToIntensity_kernel<<<(nx*ny+1023)/1024,1024>>>(d_amp,d_intensity,nx*ny);
	Normalize(d_intensity);
}
/**
 * @brief Uniformity within the region of interest
 * 
 * @param d_signal Signal whose host.uniformity is assesed
 * @param d_ROI 	Array of indexes of the elements in the ROI
 * @param n_ROI 	Length of d_ROI
 */
float OpBlocks::Uniformity(float *d_signal,unsigned int *d_ROI,unsigned int n_ROI){
	minmaxROI_kernel<<<32,1024>>>(d_signal,d_min,d_max,d_mutex,d_ROI,n_ROI);
	float h_min,h_max;
	CUDA_CALL(hipMemcpy(&h_max,d_max,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(&h_min,d_min,sizeof(float),hipMemcpyDeviceToHost));
	return 1-(h_max-h_min)/(h_max+h_min);
}
float OpBlocks::Efficiency(float *d_signal,unsigned int *d_ROI,unsigned int n_ROI,unsigned int length){
	efficiency_kernel<<<32,1024>>>(d_signal,d_min,d_max,d_mutex,d_ROI,n_ROI,length);
	float h_eff;
	CUDA_CALL(hipMemcpy(&h_eff,d_min,sizeof(float),hipMemcpyDeviceToHost));
	return h_eff;
}
float OpBlocks::Accuracy(float *d_Out,float *d_In,unsigned int *d_ROI,unsigned int n_ROI){
	float h_acc;
	accuracy_kernel<<<32,1024>>>(d_Out,d_In,d_min,d_max,d_mutex,d_ROI,n_ROI);
	CUDA_CALL(hipMemcpy(&h_acc,d_min,sizeof(float),hipMemcpyDeviceToHost));
	return h_acc;
}
void OpBlocks::PerformanceMetrics(DeviceMemory &device,HostMemory &host){
	Intensity(device.ampOut,device.intensity);
	host.uniformity.push_back(Uniformity(device.intensity,device.SR,host.n_SR));
	Normalize(device.intensity);
	Normalize(device.dint);
	host.efficiency.push_back(Efficiency(device.intensity,device.SR,host.n_SR,host.nx*host.ny));
	host.accuracy.push_back(Accuracy(device.intensity,device.dint,device.ROI,host.n_ROI));
}



//*********** PhaseRetrieve ***********//

PhaseRetrieve::PhaseRetrieve(float *gray_img,unsigned int nx,unsigned int ny, PR_Type type):
nx(nx),ny(ny){
	InitGPU(0);

	//Host memory allocation
	host.complex=(hipComplex*)malloc(nx*ny*sizeof(hipComplex));
	host.illum=(float*)malloc(nx*ny*sizeof(float));
	host.dint=(float*)malloc(nx*ny*sizeof(float));
	host.damp=(float*)malloc(nx*ny*sizeof(float));
	host.amp=(float*)malloc(nx*ny*sizeof(float));
	host.ampOut=(float*)malloc(nx*ny*sizeof(float));
	host.phSLM=(float*)malloc(nx*ny*sizeof(float));
	host.phImg=(float*)malloc(nx*ny*sizeof(float));
	host.intensity=(float*)malloc(nx*ny*sizeof(float));
	h_out_img=(float*)malloc(nx*ny*sizeof(float));
	h_out_phase=(float*)malloc(nx*ny*sizeof(float));

	//Device memory allocation
	CUDA_CALL(hipMalloc((void**)&device.complex,nx*ny*sizeof(hipComplex)));
	CUDA_CALL(hipMalloc((void**)&device.illum,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.dint,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.damp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.amp,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.ampOut,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.phSLM,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.phImg,nx*ny*sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&device.intensity,nx*ny*sizeof(float)));

	host.nx=nx;	host.ny=ny;

	SetImage(gray_img);
	SetIllumination();
	operation=new OpBlocks(nx,ny);
	SetAlgorithm(type);
}

PhaseRetrieve::~PhaseRetrieve(){
	free(host.complex);	free(host.damp);		free(host.dint);		free(host.amp);		free(host.phSLM);
	free(host.intensity);		free(h_out_img);	free(h_out_phase);	free(host.illum);
	free(host.ampOut);		free(host.phImg);
	hipFree(device.complex);	hipFree(device.damp);		hipFree(device.dint);
	hipFree(device.amp);		hipFree(device.phSLM);	
	hipFree(device.intensity);		hipFree(device.illum);
	hipFree(device.ampOut);	hipFree(device.phImg);
	if(host.ROI){	free(host.ROI);	hipFree(device.ROI);}	
	if(host.SR){	free(host.SR);	hipFree(device.SR);}
	delete algorithm;
	delete operation;
	printf("PhaseRetrieve destructed successfully!\n");
}
void PhaseRetrieve::InitGPU(int device_id){
	int devCount;
    hipGetDeviceCount(&devCount);	//number of GPUs available
	if(device_id<devCount)		//check if there are enogh GPUs
        hipSetDevice(device_id);
    else exit(1);
}
void PhaseRetrieve::SetAlgorithm(PR_Type type){
	unsigned int index=0;
	if(algorithm){
		index=algorithm->GetIndex();
		delete algorithm;
	}
	algorithm=AlgorithmCreator().FactoryMethod(operation,device,host,type);
	algorithm->SetIndex(index);
}
void PhaseRetrieve::SetImage(float *gray_img){
	float gmin=10000000.0,gmax=-10000000.0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			if(gmin>gray_img[index(i,j)])
				gmin=gray_img[index(i,j)];
			if(gmax<gray_img[index(i,j)])
				gmax=gray_img[index(i,j)];
		}
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			host.dint[index(i,j)]=(gray_img[index(i,j)]-gmin)/(gmax-gmin);
			host.damp[index(i,j)]=sqrt(host.dint[index(i,j)]);
		}
	CUDA_CALL(hipMemcpy(device.damp,host.damp,nx*ny*sizeof(float),hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(device.dint,host.dint,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(float *illum_img){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host.illum[index(i,j)]=sqrt(illum_img[index(i,j)]);
	CUDA_CALL(hipMemcpy(device.illum,host.illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::SetIllumination(){
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++)
			host.illum[index(i,j)]=sqrt(255);
	CUDA_CALL(hipMemcpy(device.illum,host.illum,nx*ny*sizeof(float),hipMemcpyHostToDevice));
}
void PhaseRetrieve::FindSR(float threshold){
	if(host.n_SR==0){
		host.SR=(unsigned int*)malloc(host.nx*host.ny*sizeof(unsigned int));
		for(unsigned int i=0;i<host.nx*host.ny;i++)
			if(host.damp[i]>threshold){
				host.SR[host.n_SR++]=i;
			}
		CUDA_CALL(hipMalloc((void**)&device.SR,host.n_SR*sizeof(unsigned int)));
		CUDA_CALL(hipMemcpy(device.SR,host.SR,host.n_SR*sizeof(unsigned int),hipMemcpyHostToDevice));
	}
}
void PhaseRetrieve::FindROI(float threshold){
	if(host.n_ROI==0){
		host.ROI=(unsigned int*)malloc(host.nx*host.ny*sizeof(unsigned int));
		for(unsigned int i=0;i<host.nx*host.ny;i++)
			if(host.damp[i]>threshold){
				host.ROI[host.n_ROI++]=i;
			}
		CUDA_CALL(hipMalloc((void**)&device.ROI,host.n_ROI*sizeof(unsigned int)));
		CUDA_CALL(hipMemcpy(device.ROI,host.ROI,host.n_ROI*sizeof(unsigned int),hipMemcpyHostToDevice));
	}
}
void PhaseRetrieve::SetROI(float x, float y, float r){
	if(host.n_ROI>0){
		hipFree(device.ROI);
		host.n_ROI=0;
	}
	else
		host.ROI=(unsigned int*)malloc(host.nx*host.ny*sizeof(unsigned int));
	int *checked;
	checked=new int[host.nx*host.ny];
	for(int i=0;i<host.nx*host.ny;i++)
		checked[i]=0;
	std::queue<int> queuex;
	std::queue<int> queuey;
	int x_pix=floor(x); int y_pix=floor(y);
	if(pow(x_pix-x,2)+pow(y_pix-y,2)<=r*r){
		queuex.push(x_pix);
		queuey.push(y_pix);
	}
	int pos[2]={-1,1};
	while(!queuex.empty()&&!queuey.empty()){
		x_pix=queuex.front(); 
		y_pix=queuey.front(); 
		queuex.pop();
		queuey.pop();
		host.ROI[host.n_ROI++]=index(x_pix,y_pix);
		checked[index(x_pix,y_pix)]=1;
		for(int i=0;i<2;i++)
			for(int j=0;j<2;j++)
				if(pow(x_pix+pos[i]-x,2)+pow(y_pix+pos[j]-y,2)<=r*r && !checked[index(x_pix,y_pix)]){
					queuex.push(x_pix+pos[i]);
					queuey.push(y_pix+pos[j]);
				}
	}
	CUDA_CALL(hipMalloc((void**)&device.ROI,host.n_ROI*sizeof(unsigned int)));
	CUDA_CALL(hipMemcpy(device.ROI,host.ROI,host.n_ROI*sizeof(unsigned int),hipMemcpyHostToDevice));
}
unsigned int PhaseRetrieve::index(unsigned int i, unsigned int j){
	return nx*i+j;
}
void PhaseRetrieve::Test(int niter){
	
	operation->RandomArray(device.phImg,-M_PI,M_PI);
	operation->RandomArray(device.phSLM,-M_PI,M_PI);

	FindSR(0.5);
	if(!host.n_ROI){
		FindROI(0.5);
		strcpy(type,"_SR");
	}
	else
		strcpy(type,"_ROI");
	algorithm->Initialize();
	
	for(int i=0;i<niter;i++){
		if(i==1) {SetAlgorithm(Weighted_GS);algorithm->Initialize();}
		algorithm->OneIteration();
		
		//if(host.n_ROI)
			//host.uniformity.push_back(operation->Uniformity(device.intensity,device.ROI,host.n_ROI));
	}

	operation->NormalizedIntensity(device.ampOut,device.intensity);

	CUDA_CALL(hipMemcpy(host.intensity,device.intensity,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.ampOut);
	CUDA_CALL(hipMemcpy(host.ampOut,device.ampOut,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.phSLM);
	CUDA_CALL(hipMemcpy(h_out_phase,device.phSLM,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	operation->Normalize(device.damp);
	CUDA_CALL(hipMemcpy(host.damp,device.damp,nx*ny*sizeof(float),hipMemcpyDeviceToHost));

	float err=0;
	for(int i=0;i<ny;i++)
		for(int j=0;j<nx;j++){
			err+=pow((host.damp[index(i,j)]-host.ampOut[index(i,j)]),2);
			h_out_img[index(i,j)]=255*host.intensity[index(i,j)];
			h_out_phase[index(i,j)]=255*h_out_phase[index(i,j)];
		}

	printf("Error squared: %f\n",err);
}

float* PhaseRetrieve::GetImage(){
	return h_out_img;
}

float* PhaseRetrieve::GetPhaseMask(){
	return h_out_phase;
}

std::vector<std::vector<float>>& PhaseRetrieve::GetMetrics(){
	metrics.push_back(host.uniformity);
	metrics.push_back(host.accuracy);
	metrics.push_back(host.efficiency);
	return metrics;
}


/********** Algorithms Implementation ***************/

void GS_ALG::OneIteration(){
	IncrementIndex();
	operation->Compose(device.complex,device.damp,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
}

void MRAF_ALG::Initialize(){
	if(index_iter==0)
		operation->ZeroArray(device.ampOut,host.nx * host.ny);
}
void MRAF_ALG::Initialize(float param){
	m=param;
	if(index_iter==0)
		operation->ZeroArray(device.ampOut,host.nx * host.ny);
}
void MRAF_ALG::OneIteration(){
	//MRAF Scaling the desired amplitude for correction
	IncrementIndex();
	operation->Normalize(device.ampOut);
	//addROI_kernel<<<(host.n_SR+1024)/1024,1024>>>(device.damp,1,device.ampOut,(m-1),device.SR,host.n_SR);
	addROI_kernel<<<(host.n_ROI+1024)/1024,1024>>>(device.damp,1,device.ampOut,(m-1),device.ROI,host.n_ROI);

	operation->Compose(device.complex,device.ampOut,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
	
	//operation->Intensity(device.ampOut,device.intensity);
}

void UCMRAF_ALG::Initialize(){
	if(index_iter==0)
		operation->ZeroArray(device.ampOut,host.nx * host.ny);
}
void UCMRAF_ALG::OneIteration(){
	//UCMRAF Scaling the desired amplitude for correction
	float u=0;
	IncrementIndex();
	if(host.uniformity.size())
		u=host.uniformity.back();
	operation->Normalize(device.ampOut);
	//addROI_kernel<<<(host.n_SR+1024)/1024,1024>>>(device.damp,1,device.ampOut,(u-1),device.SR,host.n_SR);
	addROI_kernel<<<(host.n_ROI+1024)/1024,1024>>>(device.damp,1,device.ampOut,(u-1),device.ROI,host.n_ROI);
	operation->PerformanceMetrics(device,host);
	operation->Compose(device.complex,device.ampOut,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);
	
}
void WGS_ALG::Initialize(){
	operation->ZeroArray(device.weight,host.nx*host.ny);
	CUDA_CALL(hipMemcpy(device.ampOutBefore,device.ampOut,host.nx*host.ny*sizeof(float),hipMemcpyDeviceToDevice));
}
void WGS_ALG::OneIteration(){
	//operation->Compose(device.complex,wamp,device.phImg);
	//hipMemcpy(ampOut_before,device.ampOut,host.nx*host.ny*sizeof(float),hipMemcpyDeviceToDevice);
	IncrementIndex();
	operation->Compose(device.complex,device.illum,device.phSLM);
	operation->SLM_To_Obj(device.complex,device.complex);
	operation->Decompose(device.complex,device.ampOut,device.phImg);

	operation->Intensity(device.ampOut,device.intensity);
	updatedInt();
	operation->Normalize(device.intensity);

	weight_kernel<<<(host.nx*host.ny+1023)/1024,1024>>>(device.weight,device.ampOutBefore,device.intensity,device.dint,device.ROI,host.n_ROI);
	hipDeviceSynchronize();
	operation->Normalize(device.weight);
	operation->Compose(device.complex,device.ampOut,device.phImg);
	operation->Obj_to_SLM(device.complex,device.complex);
	operation->Decompose(device.complex,device.amp,device.phSLM);
	CUDA_CALL(hipMemcpy(device.ampOutBefore,device.weight,host.nx*host.ny*sizeof(float),hipMemcpyDeviceToDevice));
}